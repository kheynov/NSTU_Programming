
#include <hip/hip_runtime.h>
#include <iostream>
using namespace std;

__global__ void add(int a, int b, int *c)
{
    *c = a + b;
}

int main()
{
    int c;
    int *dev_c;
    hipMallocManaged(&dev_c, sizeof(int));
    add<<<1, 1>>>(1, 2, dev_c);
    hipMemcpy(&c, dev_c, sizeof(int), hipMemcpyDeviceToHost);
    cout << c << endl;
    hipFree(dev_c);
    return 0;
}
