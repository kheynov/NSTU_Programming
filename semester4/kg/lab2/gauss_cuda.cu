// Libraries
#include "hip/hip_runtime.h"

#include <stdlib.h>
#include <stdio.h>
#include <time.h>

int MAX_THREADS_PER_BLOCK;

// Constants
#define startSize 5
#define maxSize 5000
#define step 5

int N = (maxSize - startSize) / step + 1; // quantity of elements

float *fillUpMatrix(int size) // Filling up matrix with elements in range [-10, 10]
{
    if (size <= 0)
        return nullptr;

    float *matrix = (float *)malloc(size * size * sizeof(float));
    int min = -10, max = 10;
    srand(time(0));

    for (int i = 0; i < size; ++i)
    {
        for (int j = 0; j < size; ++j)
        {
            matrix[i * size + j] = rand() % (max - min + 1) + min;
            if (i == j && matrix[i * size + j] == 0)
                matrix[i * size + j]++;
        }
    }

    return matrix;
}

__global__ void multiplyDetWithElement(long double *det, float *matrix, int currentDiagonalElemIndex)
{
    *det *= matrix[currentDiagonalElemIndex]; // multiplying determinant with diagonal element
}

__global__ void fillCoefsArray(float *coefs, float *matrix, int size, int currentDiagonalElemIndex, int startNumber = 0)
{
    int i = startNumber + blockDim.x * blockIdx.x + threadIdx.x;     // unique index for each coefficient
    int elemToZeroIndex = currentDiagonalElemIndex + size * (i + 1); // element that we want to cast to null

    coefs[(elemToZeroIndex / size) - 1] = -matrix[elemToZeroIndex] / matrix[currentDiagonalElemIndex];
}

__global__ void multiplyElemWithCoef(float *matrix, int size, int currentDiagonalElemRow, float *coefs, int startNumber = 0)
{
    int number = startNumber + blockDim.x * blockIdx.x + threadIdx.x;
    int columnsCount = size - currentDiagonalElemRow;

    int row = currentDiagonalElemRow + 1 + (number / columnsCount);
    int column = currentDiagonalElemRow + (number % columnsCount);

    matrix[row * size + column] += coefs[row - 1] * matrix[currentDiagonalElemRow * size + column];
}

void getNumberOfBlocksAndThreads(int elemsCount, int *blocks, int *threads)
{
    if (elemsCount < MAX_THREADS_PER_BLOCK)
    {
        *blocks = 1;
        *threads = elemsCount;
    }
    else
    {
        *blocks = elemsCount / MAX_THREADS_PER_BLOCK;
        
        if (elemsCount > *blocks * MAX_THREADS_PER_BLOCK) *blocks++;        
        
        *threads = MAX_THREADS_PER_BLOCK;

    }
}

hipError_t allocateMemory(float *matrix, float **gpuMatrix, int size, float **gpuCoefs, long double **gpuDet)
{
    // Allocating memory on GPU for determinant, matrix(1-dimension), coefficients
    hipError_t status = hipMalloc(gpuMatrix, size * size * sizeof(float));
    status = hipMemcpy(*gpuMatrix, matrix, size * size * sizeof(float), hipMemcpyHostToDevice);

    status = hipMalloc(gpuCoefs, (size - 1) * sizeof(float));

    status = hipMalloc(gpuDet, 1 * sizeof(long double));
    long double det = 1;
    status = hipMemcpy(*gpuDet, &det, 1 * sizeof(long double), hipMemcpyHostToDevice);

    return status;
}

long double gaussMethod(float *matrix, int size)
{
    long double det = 1;

    long double *_det = nullptr;
    float *_matrix = nullptr;
    float *_coefs = nullptr;

    if (allocateMemory(matrix, &_matrix, size, &_coefs, &_det) != hipSuccess)
    {
        printf("Allocate memory error!\n");
        goto freeMemory;
    }

    for (int i = 0; i < size; i++)
    {

        int curDiagonalElemIndex = i * size + i;                               // Index of current diagonal element
        multiplyDetWithElement<<<1, 1>>>(_det, _matrix, curDiagonalElemIndex); // Multiplying determinant with diagonal element

        int blocksCount, threadsCount;
        getNumberOfBlocksAndThreads(size - i - 1, &blocksCount, &threadsCount);

        fillCoefsArray<<<blocksCount, threadsCount>>>(_coefs, _matrix, size, curDiagonalElemIndex);

        int elemsCount = (size - 1 - i) * (size - i); // Elems that will be affected by iteration
        getNumberOfBlocksAndThreads(elemsCount, &blocksCount, &threadsCount);
        
        hipDeviceSynchronize(); // waiting for GPU done calculations

        multiplyElemWithCoef<<<blocksCount, threadsCount>>>(_matrix, size, i, _coefs);

        hipDeviceSynchronize(); // waiting for GPU done calculations
    }

    hipMemcpy(&det, &_det[0], sizeof(long double), hipMemcpyDeviceToHost);
    // copying memory from GPU to host

freeMemory: // freeing memory
    hipFree(_matrix);
    hipFree(_det);
    hipFree(_coefs);
    return det;
}

using namespace std;

int main()
{
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0); // getting blocks size from GPU
    MAX_THREADS_PER_BLOCK = deviceProp.maxThreadsDim[0];

    printf("Starting calculation...\n");
    for (int size = startSize, i = 0; size <= maxSize; size += step, i++)
    {
        float *matrix = fillUpMatrix(size); // filling up the matrix

        hipEvent_t start, stop;

        hipEventCreate(&start);
        hipEventCreate(&stop); // CUDA timers for calculating time

        //------
        hipEventRecord(start, 0);
        gaussMethod(matrix, size); // processing Gauss-method
        hipEventRecord(stop, 0);
        //------

        hipEventSynchronize(stop);

        float time = 0;
        hipEventElapsedTime(&time, start, stop);

        free(matrix);
        printf("%d ", size);  // matrix size
        printf("%f\n", time); // calculation time
    }
    return 0;
}
