#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdlib.h> 
#include <stdio.h>
#include <time.h>

#define MAX_THREADS_PER_BLOCK 2048


#define startSize 5
#define maxSize  5000
#define step 5

int N = (maxSize - startSize) / step + 1; //quantity of elements

using namespace std;

__global__ void multiplyDetWithElement(long double* det, float* matrix, int curDiagonalElemIndex)
{
    *det *= matrix[curDiagonalElemIndex];
}

__global__ void fillCoefsArray(float* coefs, float* matrix, int size, int curDiagonalElemIndex, int startNumber = 0)
{
    int i = startNumber + blockDim.x * blockIdx.x + threadIdx.x;
    int elemToZeroIndex = curDiagonalElemIndex + size * (i + 1);

    coefs[(elemToZeroIndex / size) - 1] = -matrix[elemToZeroIndex] / matrix[curDiagonalElemIndex];
}

__global__ void multiplyElemWithCoef(float* matrix, int size, int curDiagonalElemRow, float* coefs, int startNumber = 0)
{
    int number = startNumber + blockDim.x * blockIdx.x + threadIdx.x;
    int columnsCount = size - curDiagonalElemRow;

    int row = curDiagonalElemRow + 1 + (number / columnsCount);
    int column = curDiagonalElemRow + (number % columnsCount);

    matrix[row * size + column] += coefs[row - 1] * matrix[curDiagonalElemRow * size + column];
}

void getNumberOfBlocksAndThreads(int elemsCount, int* blocks, int* threads, int* remains)
{
    if (elemsCount < MAX_THREADS_PER_BLOCK)
    {
        *blocks = 1;
        *threads = elemsCount;
    }
    else
    {
        *blocks = elemsCount / MAX_THREADS_PER_BLOCK;
        *threads = MAX_THREADS_PER_BLOCK;
    }
    *remains = elemsCount - *blocks * *threads;
}

hipError_t allocateMemory(float* matrix, float** gpuMatrix, int size, float** gpuCoefs, long double** gpuDet)
{
    hipError_t status = hipMalloc(gpuMatrix, size * size * sizeof(float));
    status = hipMemcpy(*gpuMatrix, matrix, size * size * sizeof(float), hipMemcpyHostToDevice);

    status = hipMalloc(gpuCoefs, (size - 1) * sizeof(float));

    status = hipMalloc(gpuDet, 1 * sizeof(long double));
    long double det = 1;
    status = hipMemcpy(*gpuDet, &det, 1 * sizeof(long double), hipMemcpyHostToDevice);

    return status;
}

long double gaussMethod(float* matrix, int size)
{
    long double det = 1;

    long double* _det = nullptr;
    float* _matrix = nullptr;
    float* _coefs = nullptr;

    if (allocateMemory(matrix, &_matrix, size, &_coefs, &_det) != hipSuccess)
    {
        printf("Allocate memory error!\n");
        goto freeMemory;
    }

    for (int i = 0; i < size; i++) {

        int curDiagonalElemIndex = i * size + i;
        multiplyDetWithElement << <1, 1 >> > (_det, _matrix, curDiagonalElemIndex);


        int blocksCount, threadsCount, remains;
        getNumberOfBlocksAndThreads(size - i - 1, &blocksCount, &threadsCount, &remains);

        fillCoefsArray <<< blocksCount, threadsCount >>> (_coefs, _matrix, size, curDiagonalElemIndex);
        fillCoefsArray << < 1, remains >> > (_coefs, _matrix, size, curDiagonalElemIndex, blocksCount * threadsCount);

        int elemsCount = (size - 1 - i) * (size - i);
        getNumberOfBlocksAndThreads(elemsCount, &blocksCount, &threadsCount, &remains);

        hipDeviceSynchronize();

        multiplyElemWithCoef <<<blocksCount, threadsCount >>> (_matrix, size, i, _coefs);
        multiplyElemWithCoef << <1, remains >> > (_matrix, size, i, _coefs, blocksCount * threadsCount);

        hipDeviceSynchronize();
    }

    hipMemcpy(&det, &_det[0], sizeof(long double), hipMemcpyDeviceToHost);

    freeMemory:
    hipFree(_matrix);
    hipFree(_det);
    hipFree(_coefs);
    return det;
}

float* generateMatrix(int size)
{
    if (size <= 0)
        return nullptr;

    float* a = (float*)malloc(size * size * sizeof(float));
    int min = -10, max = 10;
    srand(time(0));

    for (int i = 0; i < size; ++i)
    {
        for (int j = 0; j < size; ++j)
        {
            a[i * size + j] = rand() % (max - min + 1) + min;
            if (i == j && a[i * size + j] == 0)
                a[i * size + j]++;
        }
    }

    return a;
}

int main()
{
    printf("Starting calculation...\n");
    for (int size = startSize, i = 0; size <= maxSize; size += step, i++)
    {
        float* matrix = generateMatrix(size);

        hipEvent_t start, stop;

        hipEventCreate(&start);
        hipEventCreate(&stop);


        //------
        hipEventRecord(start, 0);
        gaussMethod(matrix, size);
        hipEventRecord(stop, 0);
        //------

        hipEventSynchronize(stop);
        
        float time = 0;
        hipEventElapsedTime(&time, start, stop);
        // time /= 1000; //getting time in seconds

        free(matrix);
        printf("%d ", size);
        printf("%f\n", time);
    }
    return 0;
}