#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

/**
 * Сложение векторов C = A + B.
 */

#include <stdio.h>

// CUDA библиотеки
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

/**
 * Вычисляет сумму векторов А и B и записывает в вектор С. Все векторы имеют одинаковое число элементов (numElements)
 */
__global__ void vectorAdd(const float *A, const float *B, float *C,
                          int numElements) {
  int i = blockDim.x * blockIdx.x + threadIdx.x; //Индекс нити в блоке и сетке блоков

  if (i < numElements) {
    C[i] = A[i] + B[i] + 0.0f;
  }
}

int main(void) {
  // Код ошибки, которая может возникнуть во время работы
  hipError_t err = hipSuccess;

  // Вывод длины вектора, вычисление размера памяти под этот вектор
  int numElements = 50000;
  size_t size = numElements * sizeof(float);
  printf("[Vector addition of %d elements]\n", numElements);

  // Выделяем ОЗУ для вектора А
  float *h_A = (float *)malloc(size);

  // Выделяем ОЗУ для вектора В
  float *h_B = (float *)malloc(size);

  // Выделяем ОЗУ для выходного вектора С
  float *h_C = (float *)malloc(size);

  // Проверяем что выделение памяти прошло успешно
  if (h_A == NULL || h_B == NULL || h_C == NULL) {
    fprintf(stderr, "Failed to allocate host vectors!\n");
    exit(EXIT_FAILURE);
  }

  // Заполняем входные векторы A и B
  for (int i = 0; i < numElements; ++i) {
    h_A[i] = rand() / (float)RAND_MAX;
    h_B[i] = rand() / (float)RAND_MAX;
  }

  // Выделяем видеопамять под вектор A
  float *d_A = NULL;
  err = hipMalloc((void **)&d_A, size);

  if (err != hipSuccess) {
    fprintf(stderr, "Failed to allocate device vector A (error code %s)!\n",
            hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  // Выделяем видеопамять под вектор B
  float *d_B = NULL;
  err = hipMalloc((void **)&d_B, size);

  if (err != hipSuccess) {
    fprintf(stderr, "Failed to allocate device vector B (error code %s)!\n",
            hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  // Выделяем видеопамять под вектор С
  float *d_C = NULL;
  err = hipMalloc((void **)&d_C, size);

  if (err != hipSuccess) {
    fprintf(stderr, "Failed to allocate device vector C (error code %s)!\n",
            hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  // Копируем входные векторы A и B в видеопамять
  printf("Copy input data from the host memory to the CUDA device\n");
  err = hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice); //видеопамять под вектор A

  if (err != hipSuccess) {
    fprintf(stderr,
            "Failed to copy vector A from host to device (error code %s)!\n",
            hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  err = hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice); // Видеопамять под вектор В

  if (err != hipSuccess) {
    fprintf(stderr,
            "Failed to copy vector B from host to device (error code %s)!\n",
            hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  // Запускаем функцию сложения векторов на GPU
  int threadsPerBlock = 256; //Количество нитей на блок
  int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock; // Количество блоков в сетке
  printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid,
         threadsPerBlock);
  vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, numElements);
  err = hipGetLastError();

  if (err != hipSuccess) { //Вывод сообщения об ошибке если она была
    fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n",
            hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  // Копируем результат сложения векторов из видеопамяти в ОЗУ
  printf("Copy output data from the CUDA device to the host memory\n");
  err = hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

  if (err != hipSuccess) {
    fprintf(stderr,
            "Failed to copy vector C from device to host (error code %s)!\n",
            hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  // Проверяем правильность сложения
  for (int i = 0; i < numElements; ++i) {
    if (fabs(h_A[i] + h_B[i] - h_C[i]) > 1e-5) {
      fprintf(stderr, "Result verification failed at element %d!\n", i);
      exit(EXIT_FAILURE);
    }
  }

  printf("Test PASSED\n");

  // Освобождаем видеопамять для вектора А
  err = hipFree(d_A);

  if (err != hipSuccess) {
    fprintf(stderr, "Failed to free device vector A (error code %s)!\n",
            hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  err = hipFree(d_B);// Освобождаем видеопамять для вектора В

  if (err != hipSuccess) {
    fprintf(stderr, "Failed to free device vector B (error code %s)!\n",
            hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  err = hipFree(d_C);

  if (err != hipSuccess) {
    fprintf(stderr, "Failed to free device vector C (error code %s)!\n",
            hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  // Освобождаем память из ОЗУ
  free(h_A);
  free(h_B);
  free(h_C);

  printf("Done\n");
  return 0;
}
