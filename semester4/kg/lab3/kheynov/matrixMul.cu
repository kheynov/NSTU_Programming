#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

/**
 * Matrix multiplication: C = A * B.
 * Host code.
 *
 * This sample implements matrix multiplication which makes use of shared memory
 * to ensure data reuse, the matrix multiplication is done using tiling approach.
 * It has been written for clarity of exposition to illustrate various CUDA programming
 * principles, not with the goal of providing the most performant generic kernel for matrix multiplication.
 * See also:
 * V. Volkov and J. Demmel, "Benchmarking GPUs to tune dense linear algebra,"
 * in Proc. 2008 ACM/IEEE Conf. on Supercomputing (SC '08),
 * Piscataway, NJ: IEEE Press, 2008, pp. Art. 31:1-11.
 */

// Подключение системных библиотек
#include <stdio.h>
#include <assert.h>

// CUDA runtime
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

// Утилиты нужные для работы CUDA
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>

/**
 * Перемножение матриц на GPU (CUDA ядра): C = A * B
 * wA это размерность матрицы A, а wB это размерность матрицы B
 * C == результат перемножения
 */
template <int BLOCK_SIZE> __global__ void MatrixMulCUDA(float *C, float *A,
    float *B, int wA,
    int wB) {

  // Индекс блока CUDA
  int bx = blockIdx.x;
  int by = blockIdx.y;

  // Индекс Thread'a (да, я не люблю называть их "нитями")
  int tx = threadIdx.x;
  int ty = threadIdx.y;

  // Индекс первой "подматрицы" матрицы А, которая будет обрабатываться блоком
  int aBegin = wA * BLOCK_SIZE * by;

  // Индекс первой "подматрицы" матрицы А, которая будет обрабатываться блоком
  int aEnd   = aBegin + wA - 1;

  // Размер шага, с которым мы будем итерироваться по подматрицам А
  int aStep  = BLOCK_SIZE;

  // Индекс первой "подматрицы" матрицы B, которая будет обрабатываться блоком
  int bBegin = BLOCK_SIZE * bx;

  // Размер шага, с которым мы будем итерироваться по подматрицам B
  int bStep  = BLOCK_SIZE * wB;

  // Переменная Csub хранит элемент блока подматрицы, который был посчитан в thread'e
  float Csub = 0;

  // Проходим в цикле по всем подматрицам A и B
  for (int a = aBegin, b = bBegin;
       a <= aEnd;
       a += aStep, b += bStep) {

    // Выделяем массив As в общей памяти чтобы хранить подматрицу A
    __shared__ float As[BLOCK_SIZE][BLOCK_SIZE];

    // Аналогично для подматрицы А
    __shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];

    // Загружаем матрицы из видеопамяти в общую. Каждый поток загружает один элемент из каждой матрицы
    As[ty][tx] = A[a + wA * ty + tx];
    Bs[ty][tx] = B[b + wB * ty + tx];

    //Синхронизируемся чтобы убедиться что все матрицы загрузились
    __syncthreads();


    // Перемножаем матрицы, каждый поток вычисляет один элемент из блока подматрицы

#pragma unroll // Оптимизация компилятора, позволяет провернуть следующее:
      /**
        for ( int i = 0; i < 5; i++ )
          b[i] = i;
        
        =============
        
        b[0] = 0;
        b[1] = 1;
        b[2] = 2;
        b[3] = 3;
        b[4] = 4; 
       */

    for (int k = 0; k < BLOCK_SIZE; ++k) {
      Csub += As[ty][k] * Bs[k][tx]; // Собственно перемножение подматриц
    }

    // Синхронизируемся чтобы убедиться что мы всё посчитали 
    // прежде чем загружать следующие подматрицы
    __syncthreads();
  }

  // Записываем блок подматрицы в видеопамять
  // Каждый элемент отдельным thread'ом 
  int c = wB * BLOCK_SIZE * by + BLOCK_SIZE * bx;
  C[c + wB * ty + tx] = Csub;
}

void ConstantInit(float *data, int size, float val) {  // метод для заполнения массива значениями
  for (int i = 0; i < size; ++i) {
    data[i] = val;
  }
}

/**
 * Простенький тест перемножения матриц
 */
int MatrixMultiply(int argc, char **argv,
                   int block_size, const dim3 &dimsA,
                   const dim3 &dimsB) {
  
  // Выделение памяти на ОЗУ (не видеопамять) для хранения матриц А и В
  unsigned int size_A = dimsA.x * dimsA.y; //Размер матрицы А
  unsigned int mem_size_A = sizeof(float) * size_A; //Размер выделяемой памяти
  float *h_A;
  checkCudaErrors(hipHostMalloc(&h_A, mem_size_A));
  unsigned int size_B = dimsB.x * dimsB.y; // Аналогично для матрицы В
  unsigned int mem_size_B = sizeof(float) * size_B;
  float *h_B;
  checkCudaErrors(hipHostMalloc(&h_B, mem_size_B));
  hipStream_t stream;

  // Инициализируем память ОЗУ, заполняем матрицы
  const float valB = 0.01f;
  ConstantInit(h_A, size_A, 1.0f);
  ConstantInit(h_B, size_B, valB);

  // Выделяем память на GPU
  float *d_A, *d_B, *d_C;

  // Выделяем память в ОЗУ для результирующей матрицы С
  dim3 dimsC(dimsB.x, dimsA.y, 1);
  unsigned int mem_size_C = dimsC.x * dimsC.y * sizeof(float);
  float *h_C;
  checkCudaErrors(hipHostMalloc(&h_C, mem_size_C));

  if (h_C == NULL) { // Выводим ошибку если не получилось выделить память
    fprintf(stderr, "Failed to allocate host matrix C!\n");
    exit(EXIT_FAILURE);
  }

  checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_A), mem_size_A));
  checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_B), mem_size_B));
  checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_C), mem_size_C));

  // Подключаем CUDA Event'ы для подсчета времени вычисления
  hipEvent_t start, stop;
  checkCudaErrors(hipEventCreate(&start));
  checkCudaErrors(hipEventCreate(&stop));

  checkCudaErrors(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));

  // Копируем память из ОЗУ на GPU (матрицы)
  checkCudaErrors(
      hipMemcpyAsync(d_A, h_A, mem_size_A, hipMemcpyHostToDevice, stream));
  checkCudaErrors(
      hipMemcpyAsync(d_B, h_B, mem_size_B, hipMemcpyHostToDevice, stream));

  // Устанавливаем параметры запуска (блоки, сетка блоков)
  dim3 threads(block_size, block_size);
  dim3 grid(dimsB.x / threads.x, dimsA.y / threads.y);

  printf("Computing result using CUDA Kernel...\n");

  // Выполняем "разогревочные" прогоны перемножения
  if (block_size == 16) {
    MatrixMulCUDA<16>
        <<<grid, threads, 0, stream>>>(d_C, d_A, d_B, dimsA.x, dimsB.x);
  } else {
    MatrixMulCUDA<32>
        <<<grid, threads, 0, stream>>>(d_C, d_A, d_B, dimsA.x, dimsB.x);
  }

  printf("done\n");
  checkCudaErrors(hipStreamSynchronize(stream));

  // Запускаем таймер
  checkCudaErrors(hipEventRecord(start, stream));

  // Количество итераций
  int nIter = 300;

  for (int j = 0; j < nIter; j++) { //Выполняем вычисления
    if (block_size == 16) {
      MatrixMulCUDA<16>
          <<<grid, threads, 0, stream>>>(d_C, d_A, d_B, dimsA.x, dimsB.x);
    } else {
      MatrixMulCUDA<32>
          <<<grid, threads, 0, stream>>>(d_C, d_A, d_B, dimsA.x, dimsB.x);
    }
  }

  // Останавливаем таймер
  checkCudaErrors(hipEventRecord(stop, stream));

  // Ждем пока таймер остановится
  checkCudaErrors(hipEventSynchronize(stop));

  float msecTotal = 0.0f;
  checkCudaErrors(hipEventElapsedTime(&msecTotal, start, stop)); // Вычисляем затраченное время

  // Вычисляем и выводим производительность
  float msecPerMatrixMul = msecTotal / nIter;
  double flopsPerMatrixMul = 2.0 * static_cast<double>(dimsA.x) *
                             static_cast<double>(dimsA.y) *
                             static_cast<double>(dimsB.x);
  double gigaFlops =
      (flopsPerMatrixMul * 1.0e-9f) / (msecPerMatrixMul / 1000.0f);
  printf(
      "Performance= %.2f GFlop/s, Time= %.3f msec, Size= %.0f Ops,"
      " WorkgroupSize= %u threads/block\n",
      gigaFlops, msecPerMatrixMul, flopsPerMatrixMul, threads.x * threads.y);

  // Копируем матрицу с результатом вычисления с GPU на ОЗУ
  checkCudaErrors(
      hipMemcpyAsync(h_C, d_C, mem_size_C, hipMemcpyDeviceToHost, stream));
  checkCudaErrors(hipStreamSynchronize(stream));

  printf("Checking computed result for correctness: ");
  bool correct = true;

  // Вычисляем ошибку по формуле
  //     |<x, y>_cpu - <x,y>_gpu|/<|x|, |y|>  < eps
  double eps = 1.e-6;  // Машинный "ноль"

  for (int i = 0; i < static_cast<int>(dimsC.x * dimsC.y); i++) {
    double abs_err = fabs(h_C[i] - (dimsA.x * valB));
    double dot_length = dimsA.x;
    double abs_val = fabs(h_C[i]);
    double rel_err = abs_err / abs_val / dot_length;

    if (rel_err > eps) {
      printf("Error! Matrix[%05d]=%.8f, ref=%.8f error term is > %E\n",
             i, h_C[i], dimsA.x * valB, eps);
      correct = false;
    }
  }

  printf("%s\n", correct ? "Result = PASS" : "Result = FAIL");

  // Освобождаем память
  checkCudaErrors(hipHostFree(h_A));
  checkCudaErrors(hipHostFree(h_B));
  checkCudaErrors(hipHostFree(h_C));
  checkCudaErrors(hipFree(d_A));
  checkCudaErrors(hipFree(d_B));
  checkCudaErrors(hipFree(d_C));
  checkCudaErrors(hipEventDestroy(start));
  checkCudaErrors(hipEventDestroy(stop));
  printf(
      "\nNOTE: The CUDA Samples are not meant for performance "
      "measurements. Results may vary when GPU Boost is enabled.\n");

  if (correct) {
    return EXIT_SUCCESS;
  } else {
    return EXIT_FAILURE;
  }
}


/**
 * Программа main
 */
int main(int argc, char **argv) {
  printf("[Matrix Multiply Using CUDA] - Starting...\n");

  if (checkCmdLineFlag(argc, (const char **)argv, "help") ||
      checkCmdLineFlag(argc, (const char **)argv, "?")) {
    printf("Usage -device=n (n >= 0 for deviceID)\n");
    printf("      -wA=WidthA -hA=HeightA (Width x Height of Matrix A)\n");
    printf("      -wB=WidthB -hB=HeightB (Width x Height of Matrix B)\n");
    printf("  Note: Outer matrix dimensions of A & B matrices" \
           " must be equal.\n");

    exit(EXIT_SUCCESS);
  }

  // Эта функция выберет лучший из доступных видеоадаптеров
  int dev = findCudaDevice(argc, (const char **)argv);

  int block_size = 32;

  dim3 dimsA(5 * 2 * block_size, 5 * 2 * block_size, 1);
  dim3 dimsB(5 * 4 * block_size, 5 * 2 * block_size, 1);

  // Ширина матрицы А
  if (checkCmdLineFlag(argc, (const char **)argv, "wA")) {
    dimsA.x = getCmdLineArgumentInt(argc, (const char **)argv, "wA");
  }

  // Высота матрицы А
  if (checkCmdLineFlag(argc, (const char **)argv, "hA")) {
    dimsA.y = getCmdLineArgumentInt(argc, (const char **)argv, "hA");
  }

  // Ширина матрицы В
  if (checkCmdLineFlag(argc, (const char **)argv, "wB")) {
    dimsB.x = getCmdLineArgumentInt(argc, (const char **)argv, "wB");
  }

  // Высота матрицы В
  if (checkCmdLineFlag(argc, (const char **)argv, "hB")) {
    dimsB.y = getCmdLineArgumentInt(argc, (const char **)argv, "hB");
  }

  if (dimsA.x != dimsB.y) { //Если ширина матрицы А и высота матрицы В не совпадает, выдаем ошибку, мы не сможем 
  // перемножить такие матрицы
    printf("Error: outer matrix dimensions must be equal. (%d != %d)\n",
           dimsA.x, dimsB.y);
    exit(EXIT_FAILURE);
  }

  printf("MatrixA(%d,%d), MatrixB(%d,%d)\n", dimsA.x, dimsA.y,
         dimsB.x, dimsB.y);

  checkCudaErrors(hipProfilerStart()); //запускаем измеритель производительности
  int matrix_result = MatrixMultiply(argc, argv, block_size, dimsA, dimsB); //Выполняем вычисления
  checkCudaErrors(hipProfilerStop()); //Останавливаем измеритель производительности

  exit(matrix_result);
}
