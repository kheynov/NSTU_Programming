#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

/*
    This example demonstrates how to use the Cuda OpenGL bindings to
    dynamically modify a vertex buffer using a Cuda kernel.

    The steps are:
    1. Create an empty vertex buffer object (VBO)
    2. Register the VBO with Cuda
    3. Map the VBO for writing from Cuda
    4. Run Cuda kernel to modify the vertex positions
    5. Unmap the VBO
    6. Render the results using OpenGL

    Host code
*/

// подключение библиотек
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

// OpenGL
#include <helper_gl.h>

#include <GL/freeglut.h>

// CUDA
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>
#include <helper_functions.h>    //  hip/hip_runtime.h, hip/hip_runtime_api.h
#include <hip/hip_runtime_api.h>        

#include <hip/hip_vector_types.h>

#define MAX_EPSILON_ERROR 10.0f
#define THRESHOLD          0.30f
#define REFRESH_DELAY     5 //ms

// константы
const unsigned int window_width  = 800;
const unsigned int window_height = 600;

const unsigned int mesh_width    = 256;
const unsigned int mesh_height   = 256;

// Типы данных для буффера точек
GLuint vbo;
struct hipGraphicsResource *cuda_vbo_resource;
void *d_vbo_buffer = NULL;

float g_fAnim = 0.0;

// для вращения камеры мышью
int mouse_old_x, mouse_old_y;
int mouse_buttons = 0;
float rotate_x = 0.0, rotate_y = 0.0;
float translate_z = -3.0;

StopWatchInterface *timer = NULL;


int fpsCount = 0;   
int fpsLimit = 1;        
int g_Index = 0;
float avgFPS = 0.0f;
unsigned int frameCount = 0;
unsigned int g_TotalErrors = 0;
bool g_bQAReadback = false;

int *pArgc = NULL;
char **pArgv = NULL;

#define MAX(a,b) ((a > b) ? a : b)

bool runTest(int argc, char **argv, char *ref_file);
void cleanup();

// GL функции
bool initGL(int *argc, char **argv);
void createVBO(GLuint *vbo, struct hipGraphicsResource **vbo_res,
               unsigned int vbo_res_flags);
void deleteVBO(GLuint *vbo, struct hipGraphicsResource *vbo_res);

// callbacks
void display();
void keyboard(unsigned char key, int x, int y);
void mouse(int button, int state, int x, int y);
void motion(int x, int y);
void timerEvent(int value);

// Cuda функции
void runCuda(struct hipGraphicsResource **vbo_resource);
void runAutoTest(int devID, char **argv, char *ref_file);

const char *sSDKsample = "RGZ (Paraboloid translation)";

__global__ void simple_vbo_kernel(float4 *pos, unsigned int width, unsigned int height, float time)
{
    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

    // вычисление координат
    float u = x / (float) width;
    float v = y / (float) height;
    u = u*2.0f - 1.0f;
    v = v*2.0f - 1.0f;

    // вычисление z-координаты в точке
    float coef = 2.0f;
    float w = (u*u)/(coef) * sinf(time)*-1.0f + ((v*v)/(coef) * sinf(time));
    
    //итоговая вершина
    pos[y*width+x] = make_float4(u, w, v, 1.0f);
}


void launch_kernel(float4 *pos, unsigned int mesh_width,
                   unsigned int mesh_height, float time)
{
    //запуск вычисления
    dim3 block(8, 8, 1);
    dim3 grid(mesh_width / block.x, mesh_height / block.y, 1);
    simple_vbo_kernel<<< grid, block>>>(pos, mesh_width, mesh_height, time);
}


//точка входа в программу
int main(int argc, char **argv)
{
    char *ref_file = NULL;

    pArgc = &argc;
    pArgv = argv;

    setenv ("DISPLAY", ":0", 0);

    printf("%s starting...\n", sSDKsample);

    if (argc > 1)
    {
        if (checkCmdLineFlag(argc, (const char **)argv, "file"))
        {
            getCmdLineArgumentString(argc, (const char **)argv, "file", (char **)&ref_file);
        }
    }

    printf("\n");

    runTest(argc, argv, ref_file);

    printf("%s completed, returned %s\n", sSDKsample, (g_TotalErrors == 0) ? "OK" : "ERROR!");
    exit(g_TotalErrors == 0 ? EXIT_SUCCESS : EXIT_FAILURE);
}

void computeFPS()
{
    frameCount++;
    fpsCount++;

    if (fpsCount == fpsLimit)
    {
        avgFPS = 1.f / (sdkGetAverageTimerValue(&timer) / 1000.f);
        fpsCount = 0;
        fpsLimit = (int)MAX(avgFPS, 1.f);

        sdkResetTimer(&timer);
    }

    char fps[256];
    sprintf(fps, "Cuda GL Interop (VBO): %3.1f fps (Max 100Hz)", avgFPS);
    glutSetWindowTitle(fps);
}

// инициализация openGL
bool initGL(int *argc, char **argv)
{
    glutInit(argc, argv);
    glutInitDisplayMode(GLUT_RGBA | GLUT_DOUBLE);
    glutInitWindowSize(window_width, window_height);
    glutCreateWindow("Cuda GL Interop (VBO)");
    glutDisplayFunc(display);
    glutKeyboardFunc(keyboard);
    glutMotionFunc(motion);
    glutTimerFunc(REFRESH_DELAY, timerEvent,0);

    // инициализируем необходимые GL библиотеки
    if (! isGLVersionSupported(2,0))
    {
        fprintf(stderr, "ERROR: Support for necessary OpenGL extensions missing.");
        fflush(stderr);
        return false;
    }

    glClearColor(0.0, 0.0, 0.0, 1.0);
    glDisable(GL_DEPTH_TEST);

    // размер окна
    glViewport(0, 0, window_width, window_height);

    // проекция(камера)
    glMatrixMode(GL_PROJECTION);
    glLoadIdentity();
    gluPerspective(60.0, (GLfloat)window_width / (GLfloat) window_height, 0.1, 10.0);

    SDK_CHECK_ERROR_GL();

    return true;
}


bool runTest(int argc, char **argv, char *ref_file)
{
    //  Создаём таймер
    sdkCreateTimer(&timer);

    // выбираем предпочтительный видеоадаптер
    int devID = findCudaDevice(argc, (const char **)argv);

    if (ref_file != NULL)
    {
        // регистрируем в видеопамяти буффер для хранения вершин
        checkCudaErrors(hipMalloc((void **)&d_vbo_buffer, mesh_width*mesh_height*4*sizeof(float)));

        // запускаем вычисление на GPU
        runAutoTest(devID, argv, ref_file);

        hipFree(d_vbo_buffer); //освобождаем память
        d_vbo_buffer = NULL;
    }
    else
    {
        // Проверяем правильно ли инициализировалась GL
        if (false == initGL(&argc, argv))
        {
            return false;
        }

        // регистрируем колбеки
        glutDisplayFunc(display);
        glutKeyboardFunc(keyboard);
        glutMouseFunc(mouse);
        glutMotionFunc(motion);
        glutCloseFunc(cleanup);

        // создаем буффер вершин
        createVBO(&vbo, &cuda_vbo_resource, cudaGraphicsMapFlagsWriteDiscard);

        // запускаем часть вычислений на CUDA
        runCuda(&cuda_vbo_resource);

        // запускаем главный цикл рендера
        glutMainLoop();
    }

    return true;
}


void runCuda(struct hipGraphicsResource **vbo_resource)
{
    // map OpenGL buffer object for writing from CUDA
    // привязываем объекты OpenGL буффера для того чтобы CUDA могла записывать в них информацию
    float4 *dptr;
    checkCudaErrors(hipGraphicsMapResources(1, vbo_resource, 0));
    size_t num_bytes;
    checkCudaErrors(hipGraphicsResourceGetMappedPointer((void **)&dptr, &num_bytes,
                                                         *vbo_resource));
    launch_kernel(dptr, mesh_width, mesh_height, g_fAnim);

    // отвязываем буффер от CUDA
    checkCudaErrors(hipGraphicsUnmapResources(1, vbo_resource, 0));
}


#ifndef FOPEN
#define FOPEN(fHandle,filename,mode) (fHandle = fopen(filename, mode))
#endif


void sdkDumpBin2(void *data, unsigned int bytes, const char *filename)
{
    printf("sdkDumpBin: <%s>\n", filename);
    FILE *fp;
    FOPEN(fp, filename, "wb");
    fwrite(data, bytes, 1, fp);
    fflush(fp);
    fclose(fp);
}

void runAutoTest(int devID, char **argv, char *ref_file)
{
    char *reference_file = NULL;
    void *imageData = malloc(mesh_width*mesh_height*sizeof(float));

    // запуск вычислений
    launch_kernel((float4 *)d_vbo_buffer, mesh_width, mesh_height, g_fAnim);

    hipDeviceSynchronize();
    getLastCudaError("launch_kernel failed");

    checkCudaErrors(hipMemcpy(imageData, d_vbo_buffer, mesh_width*mesh_height*sizeof(float), hipMemcpyDeviceToHost));

    sdkDumpBin2(imageData, mesh_width*mesh_height*sizeof(float), "simpleGL.bin");
    reference_file = sdkFindFilePath(ref_file, argv[0]);

    if (reference_file &&
        !sdkCompareBin2BinFloat("simpleGL.bin", reference_file,
                                mesh_width*mesh_height*sizeof(float),
                                MAX_EPSILON_ERROR, THRESHOLD, pArgv[0]))
    {
        g_TotalErrors++;
    }
}

// функция создающая буффер точек
void createVBO(GLuint *vbo, struct hipGraphicsResource **vbo_res,
               unsigned int vbo_res_flags)
{
    assert(vbo);

    // создание объекта буффера
    glGenBuffers(1, vbo);
    glBindBuffer(GL_ARRAY_BUFFER, *vbo);

    unsigned int size = mesh_width * mesh_height * 4 * sizeof(float);
    glBufferData(GL_ARRAY_BUFFER, size, 0, GL_DYNAMIC_DRAW);
    
    glBindBuffer(GL_ARRAY_BUFFER, 0);

    // Регистрируем буффер в CUDA
    checkCudaErrors(hipGraphicsGLRegisterBuffer(vbo_res, *vbo, vbo_res_flags));

    SDK_CHECK_ERROR_GL();
}

//удаляем буффер
void deleteVBO(GLuint *vbo, struct hipGraphicsResource *vbo_res)
{

    // unregister this buffer object with CUDA
    checkCudaErrors(hipGraphicsUnregisterResource(vbo_res));

    glBindBuffer(1, *vbo);
    glDeleteBuffers(1, vbo);

    *vbo = 0;
}

//Колбек вызывающийся при обновлении экрана
void display()
{
    sdkStartTimer(&timer);

    // run CUDA kernel to generate vertex positions
    // запускаем CUDA ядра чтобы сгенерировать позиции вершин
    runCuda(&cuda_vbo_resource);

    glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT); //очищаем дисплей

    // отображаем окно просмотра (камеру)
    glMatrixMode(GL_MODELVIEW);
    glLoadIdentity();
    glTranslatef(0.0, 0.0, translate_z);
    glRotatef(rotate_x, 1.0, 0.0, 0.0);
    glRotatef(rotate_y, 0.0, 1.0, 0.0);

    // рендерим буффер
    glBindBuffer(GL_ARRAY_BUFFER, vbo);
    glVertexPointer(4, GL_FLOAT, 0, 0);

    glEnableClientState(GL_VERTEX_ARRAY);
    
    glColor3f(1.0, 1.0, 0.0); // делаем поверхность жёлтой
    glDrawArrays(GL_POINTS, 0, mesh_width * mesh_height);
    glDisableClientState(GL_VERTEX_ARRAY);

    glutSwapBuffers();

    g_fAnim += 0.01f;

    sdkStopTimer(&timer);
    computeFPS();
}

void timerEvent(int value)
{
    if (glutGetWindow())
    {
        glutPostRedisplay();
        glutTimerFunc(REFRESH_DELAY, timerEvent,0);
    }
}

void cleanup()
{
    sdkDeleteTimer(&timer);

    if (vbo)
    {
        deleteVBO(&vbo, cuda_vbo_resource);
    }
}


//обработчик нажатий на клавиатуру (клавиша ESC)
void keyboard(unsigned char key, int /*x*/, int /*y*/)
{
    switch (key)
    {
        case (27) :
                glutDestroyWindow(glutGetWindow());
                return;
    }
}

// обработчик событий мыши
void mouse(int button, int state, int x, int y)
{
    if (state == GLUT_DOWN)
    {
        mouse_buttons |= 1<<button;
    }
    else if (state == GLUT_UP)
    {
        mouse_buttons = 0;
    }

    mouse_old_x = x;
    mouse_old_y = y;
}

void motion(int x, int y)
{
    float dx, dy;
    dx = (float)(x - mouse_old_x);
    dy = (float)(y - mouse_old_y);

    if (mouse_buttons & 1)
    {
        rotate_x += dy * 0.2f;
        rotate_y += dx * 0.2f;
    }
    else if (mouse_buttons & 4)
    {
        translate_z += dy * 0.01f;
    }

    mouse_old_x = x;
    mouse_old_y = y;
}
